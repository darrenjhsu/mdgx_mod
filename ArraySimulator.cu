#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <nvml.h>
#include <hiprand/hiprand_kernel.h>

#include "MultiSimDS.h"
#include "KernelMacros.h"

__device__ __constant__ gpuMultiSim cGms;

//-----------------------------------------------------------------------------
// GetGpuSpecs: obtain specs for the GPU in the system.  This is here to have
//              all of the CUDA-specific data structures in this one unit,
//              while returning the information that's really needed in a
//              custom struct full of POD types to be incorporated into the
//              main peptide control data structure in Peptide.c.
//
// Arguments:
//   reckless:   taken from the trajectory data structure and fed in from the
//               command line, lets mdgx go ahead and run on GPUs that are
//               already in use
//-----------------------------------------------------------------------------
extern "C" gpuSpecs GetGpuSpecs(int reckless)
{
  int i, ndev, gpucount, seldev;
  unsigned int nvmlItemCount;
  int* validGpus;
  int* gpuList;
  hipError_t stt;
  nvmlReturn_t sttNV;
  hipDeviceProp_t devPRP;
  gpuSpecs devspc;
  nvmlProcessInfo_t nvmlInfo[32];
  nvmlDevice_t ntdev;
  
  // Test that there is a GPU in the system
  stt = hipGetDeviceCount(&gpucount);
  if (gpucount == 0) {
    printf("mdgx >> Error.  No CUDA-capable devices were found.\n");
    hipDeviceReset();
    exit(1);
  }

  // Activate zero-copy
  hipSetDeviceFlags(hipDeviceMapHost);

  // Initialize the NVIDIA Management Library
  nvmlInit();
  
  // Get device properties
  validGpus = (int*)malloc(gpucount * sizeof(int));
  gpuList = (int*)malloc(gpucount * sizeof(int));
  ndev = 0;
  for (i = 0; i < gpucount; i++) {
    hipGetDeviceProperties(&devPRP, i);
    if (devPRP.major >= 3) {
      nvmlDeviceGetHandleByIndex(i, &ntdev);
      nvmlItemCount = 0;
      sttNV = nvmlDeviceGetComputeRunningProcesses(ntdev, &nvmlItemCount,
                                                   nvmlInfo);
      if (sttNV != NVML_SUCCESS && sttNV != NVML_ERROR_INSUFFICIENT_SIZE) {
	printf("mdgx >> Warning.  Unable to monitor activity on GPU %d "
	       "[error %u]\n", i, sttNV);
      }
      if (nvmlItemCount == 0 || reckless == 1) {
        validGpus[i] = 1;
        gpuList[ndev] = i;
        ndev++;
      }
    }
  }
  if (ndev == 0 && reckless == 0) {
    printf("mdgx >> All GPUs are unavailable, or assisting other customers.  "
           "If you believe\nmdgx >> you have received this message in error, "
	   "or if you know people are\nmdgx >> using the GPUs and just don't "
	   "care, you may re-run mdgx with the\nmdgx >> -Reckless flag.  What "
           "your colleagues say at the water cooler is\nmdgx >> not the "
	   "responsibility of Amber developers.\n");
    exit(1);
  }

  // Shut down the NVIDIA Management Lbirary
  nvmlShutdown();
  
  // Select a device from the list
  stt = hipSetValidDevices(gpuList, ndev);
  if (stt != hipSuccess) {
    printf("mdgx >> Error searching for CUDA-compatible GPU.\n");
    hipDeviceReset();
    exit(1);
  }

  // Establish the CUDA context
  stt = hipFree(0);
  if (stt != hipSuccess) {
    printf("mdgx >> Error selecting compatible GPU.\n");
    hipDeviceReset();
    exit(1);
  }

  // Get the device
  stt = hipGetDevice(&seldev);
  if (stt != hipSuccess) {
    printf("mdgx >> Error setting cuda device.\n");
    hipDeviceReset();
    exit(1);
  }
  hipDeviceSynchronize();
  hipGetDeviceProperties(&devPRP, seldev);

  // Copy the relevant information for shipment back to the calling function
  devspc.major          = devPRP.major;
  devspc.minor          = devPRP.minor;
  devspc.MPcount        = devPRP.multiProcessorCount;
  devspc.maxThrPerMP    = devPRP.maxThreadsPerMultiProcessor;
  devspc.maxThrPerBlock = devPRP.maxThreadsPerBlock;
  devspc.cardMemory     = devPRP.totalGlobalMem;
  i = strlen(devPRP.name);
  if (i > 127) {
    i = 127;
  }
  strncpy(devspc.name, devPRP.name, i);
  devspc.name[i] = '\0';
  
  // Free allocated memory
  free(gpuList);
  free(validGpus);

  return devspc;
}

//-----------------------------------------------------------------------------
// kGpuPRNGSetup: kernel for initializing GPU random number generators
//-----------------------------------------------------------------------------
__global__ void kGpuPRNGSetup(hiprandState_t *states, int igseed)
{
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  hiprand_init(igseed, tid, 0, &states[tid]);
}

//-----------------------------------------------------------------------------
// InitGpuPRNG: initialize pseudo-random number generators on the GPU.
//
// Arguments:
//   gms:       the repository for all parameters amd coordinates
//   igseed:    the random number generator seed
//   nblocks:   the number of blocks that the main dynamics kernels will run
//   blockDim:  dimension of the main dynamics kernel blocks
//-----------------------------------------------------------------------------
extern "C" void InitGpuPRNG(gpuMultiSim *gms, int igseed, int nblocks,
			    int blockdim)
{
  hipMalloc((void **)&gms->prngStates,
             nblocks * blockdim * sizeof(hiprandState));
  kGpuPRNGSetup<<<nblocks, blockdim>>>((hiprandState_t*)gms->prngStates,
                                       igseed);
}

//-----------------------------------------------------------------------------
// SetGmsImage: function to establish a GPU Multi-Simulator on the device,
//              with pointers to all of the device-allocated memory as well as
//              constants describing the simulation conditions.
//
// Arguments:
//   gms:    the repository for all parameters amd coordinates
//-----------------------------------------------------------------------------
extern "C" void SetGmsImage(gpuMultiSim *gms)
{
  hipError_t status;

  status = hipMemcpyToSymbol(HIP_SYMBOL(cGms), gms, sizeof(gpuMultiSim));
  if (status != hipSuccess) {
    printf("SetGmsImage >> Unable to copy gpuMultiSim struct to the "
           "device (error %d).\n", (int)status);
    exit(1);
  }
}

//----------------------------------------------------------------------------
// kSetSystemCounters: set counters to guide the blocks as they step through
//                     systems during each segment of dynamics, in between
//                     coordinate writes.
//-----------------------------------------------------------------------------
__global__ void kSetSystemCounters(int blocks)
{
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  while (tid < 2 * cGms.nsgmdout) {
    cGms.DVCsystemPos[tid] = blocks;
    tid += gridDim.x * blockDim.x;
  }
}

//-----------------------------------------------------------------------------
// Dynamics kernels with RATTLE to compute forces (kDynLoop) or force and
// energies (kEStep).  The concept is that kEStep will launch to get forces and
// energies for the first step, then kDynLoop will fire off for (ntpr - 1)
// steps, then kEStep will launch again, and so on until the maximum number of
// steps has been reached.
//-----------------------------------------------------------------------------
#define GO_RATTLE
#define ATOM_LIMIT SM_ATOM_COUNT
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750)
#  define THREAD_COUNT 256
#else
#  define THREAD_COUNT 288
#endif
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kEStepSmallRtt(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kDynLoopSmallRtt(int sgc)
#include "kDynamics.h"
#define GBSOLVENT
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kEStepSmallGBRtt(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kDynLoopSmallGBRtt(int sgc)
#include "kDynamics.h"
#undef GBSOLVENT
#undef THREAD_COUNT
#undef ATOM_LIMIT

#define ATOM_LIMIT MD_ATOM_COUNT
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750)
#  define THREAD_COUNT 512
#else
#  define THREAD_COUNT 576
#endif
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kEStepMedRtt(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kDynLoopMedRtt(int sgc)
#include "kDynamics.h"
#define GBSOLVENT
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kEStepMedGBRtt(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kDynLoopMedGBRtt(int sgc)
#include "kDynamics.h"
#undef GBSOLVENT
#undef THREAD_COUNT
#undef ATOM_LIMIT

#define ATOM_LIMIT LG_ATOM_COUNT
#define THREAD_COUNT 1024
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kEStepLargeRtt(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kDynLoopLargeRtt(int sgc)
#include "kDynamics.h"
#define GBSOLVENT
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kEStepLargeGBRtt(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kDynLoopLargeGBRtt(int sgc)
#include "kDynamics.h"
#undef GBSOLVENT
#undef THREAD_COUNT
#undef ATOM_LIMIT
#undef GO_RATTLE

//-----------------------------------------------------------------------------
// Kernels without RATTLE.  The concept behind DynLoop and EStep kernels is the
// same as above, but without the register burden of RATTLE the kernels can
// engage additional threads.
//-----------------------------------------------------------------------------
#define ATOM_LIMIT SM_ATOM_COUNT
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750)
#  define THREAD_COUNT 256
#else
#  define THREAD_COUNT 320
#endif
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kEStepSmall(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kDynLoopSmall(int sgc)
#include "kDynamics.h"
#define GBSOLVENT
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kEStepSmallGB(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 4) kDynLoopSmallGB(int sgc)
#include "kDynamics.h"
#undef GBSOLVENT
#undef THREAD_COUNT
#undef ATOM_LIMIT

#define ATOM_LIMIT MD_ATOM_COUNT
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750)
#  define THREAD_COUNT 512
#else
#  define THREAD_COUNT 640
#endif
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kEStepMed(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kDynLoopMed(int sgc)
#include "kDynamics.h"
#define GBSOLVENT
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kEStepMedGB(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 2) kDynLoopMedGB(int sgc)
#include "kDynamics.h"
#undef GBSOLVENT
#undef THREAD_COUNT
#undef ATOM_LIMIT

#define ATOM_LIMIT LG_ATOM_COUNT
#define THREAD_COUNT 1024
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kEStepLarge(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kDynLoopLarge(int sgc)
#include "kDynamics.h"
#define GBSOLVENT
#define COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kEStepLargeGB(int sgc)
#include "kDynamics.h"
#undef COMPUTE_ENERGY
__global__ void __launch_bounds__(THREAD_COUNT, 1) kDynLoopLargeGB(int sgc)
#include "kDynamics.h"
#undef GBSOLVENT
#undef THREAD_COUNT
#undef ATOM_LIMIT
  
//-----------------------------------------------------------------------------
// LaunchDynamics: launch the appropriate kernels for energy and forces.  As
//                 is done in the pmemd code, this function in the CUDA unit
//                 encapsualtes the launch so that the .c libraries can be
//                 built with a standard C compiler.
//
// Arguments:
//   gms:       the repository for all parameters amd coordinates
//   blockDim:  the block size to use, determined by PlanGpuUtilization in
//              Peptide.c
//   devspc:    device specifications
//-----------------------------------------------------------------------------
extern "C" void LaunchDynamics(gpuMultiSim *gms, int blockDim, int nblocks,
			       gpuSpecs *devspc)
{
  int i;

  // Initialize system counters for this portion of dynamics
  kSetSystemCounters<<<nblocks, blockDim>>>(nblocks);
  
  // Vacuum-phase dynamics
  if (gms->igb == 6) {
    if (gms->rattle == 0) {
      for (i = 0; i < gms->nsgmdout; i++) {
        if (blockDim < 512) {
          kEStepSmall<<<nblocks, blockDim>>>(i);
          kDynLoopSmall<<<nblocks, blockDim>>>(i);
        }
        else if (blockDim < 1024) {
          kEStepMed<<<nblocks, blockDim>>>(i);
          kDynLoopMed<<<nblocks, blockDim>>>(i);
        }
        else {
          kEStepLarge<<<nblocks, blockDim>>>(i);
          kDynLoopLarge<<<nblocks, blockDim>>>(i);
        }
      }
    }
    else {
      for (i = 0; i < gms->nsgmdout; i++) {
        if (blockDim < 512) {
          kEStepSmallRtt<<<nblocks, blockDim>>>(i);
          kDynLoopSmallRtt<<<nblocks, blockDim>>>(i);
        }
        else if (blockDim < 1024) {
          kEStepMedRtt<<<nblocks, blockDim>>>(i);
          kDynLoopMedRtt<<<nblocks, blockDim>>>(i);
        }
        else {
          kEStepLargeRtt<<<nblocks, blockDim>>>(i);
          kDynLoopLargeRtt<<<nblocks, blockDim>>>(i);
        }
      }
    }
  }

  // Dynamics in Generalized Born solvent
  else {
    if (gms->rattle == 0) {
      for (i = 0; i < gms->nsgmdout; i++) {
        if (blockDim < 512) {
          kEStepSmallGB<<<nblocks, blockDim>>>(i);
          kDynLoopSmallGB<<<nblocks, blockDim>>>(i);
        }
        else if (blockDim < 1024) {
          kEStepMedGB<<<nblocks, blockDim>>>(i);
          kDynLoopMedGB<<<nblocks, blockDim>>>(i);
        }
        else {
          kEStepLargeGB<<<nblocks, blockDim>>>(i);
          kDynLoopLargeGB<<<nblocks, blockDim>>>(i);
	}
      }
    }
    else {
      for (i = 0; i < gms->nsgmdout; i++) {
        if (blockDim < 512) {
          kEStepSmallGBRtt<<<nblocks, blockDim>>>(i);
          kDynLoopSmallGBRtt<<<nblocks, blockDim>>>(i);
        }
        else if (blockDim < 1024) {
          kEStepMedGBRtt<<<nblocks, blockDim>>>(i);
          kDynLoopMedGBRtt<<<nblocks, blockDim>>>(i);
        }
        else {
          kEStepLargeGBRtt<<<nblocks, blockDim>>>(i);
          kDynLoopLargeGBRtt<<<nblocks, blockDim>>>(i);
	}
      }
    }
  }
}
